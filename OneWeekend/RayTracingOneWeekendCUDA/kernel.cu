﻿#include <iostream>
#include <fstream>
#include "hip/hip_runtime.h"


#include <stdio.h>

#define N 256

struct colors
{
    int r[N], g[N], b[N];

    void store(std::ostream& out)
    {
        for (int i = 0; i < N; i++)
        {
            out << r[i] << " " << g[i] << " " << b[i] << std::endl;
        }
        
    }
};

__global__ void render(std::ostream& out, int* width, int* height, colors* color)
{
    for (int j = 0; j < *height; ++j) {
        for (int i = 0; i < *width; ++i) {
            auto r = double(i) / (*width - 1);
            auto g = double(j) / (*height - 1);
            auto b = 0;

            color->r[N] = static_cast<int>(255.999 * r);
            color->g[N] = static_cast<int>(255.999 * g);
            color->b[N] = static_cast<int>(255.999 * b);

        }
    }
}

int main()
{
    std::ofstream myFile;
    myFile.open("image.ppm");
    // Image

    int image_width = 256;
    int image_height = 256;

    int* width, *height, *out;
    int* cu_width, *cu_height, *cu_out;
    /**
    *  Allocate memory to CPU vars first.
    *  Create GPU mem vars.
    *  Allocate memory to GPU mems.
    *  Then copy CPU mem in them.
    *  Make sure to create an output var that returns the mem from GPU to CPU
    * 
    */

    // CPU pointers
    width =     (int*)malloc(sizeof(float) * N);
    height =    (int*)malloc(sizeof(float) * N);
    out =       (int*)malloc(sizeof(float) * N);

    // GPU pointers
    hipMalloc((void**)&cu_width,   sizeof(int) * N);
    hipMalloc((void**)&cu_height,  sizeof(int) * N);
    hipMalloc((void**)&cu_out,     sizeof(int) * N);

    // Copy CPU memory into GPU memory
    hipMemcpy(cu_width, width, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(cu_height, height, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(cu_out, out, sizeof(int) * N, hipMemcpyHostToDevice);

    // Render
    myFile << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    colors* color = new colors[sizeof(int) * N];

    render<<<1,256>>>(myFile, cu_width, cu_height, color);
    color->store(myFile);

        
        //store(myFile);

    //myFile << color.r << " " << color.g << " " << color.b << std::endl;


    myFile.close();

    hipFree(cu_width);
    hipFree(cu_height);
    hipFree(cu_out);

    free(width);
    free(height);
    free(out);

    return 0;

}